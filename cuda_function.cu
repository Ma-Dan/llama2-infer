#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <string.h>
#include <fcntl.h>
#if defined _WIN32
    #include "win.h"
#else
    #include <unistd.h>
    #include <sys/mman.h>
#endif

#include <hip/hip_runtime_api.h>
#include <hip/hip_fp16.h>
#include <hipcub/hipcub.hpp>
#include <hipblas.h>

hipblasHandle_t handle;

void createCublas()
{
    hipblasCreate(&handle);
}

void destroyCublas()
{
    hipblasDestroy(handle);
}

int mallocGPUData(float** gpuData, int length)
{
    hipMalloc(gpuData, length);
    return 0;
}

int uploadGPUData(void *scratchGpu, void *scratchCpu, int length)
{
    hipMemcpyAsync(scratchGpu, scratchCpu, length, hipMemcpyHostToDevice);
    return 0;
}

void freeGPUData(void *gpuData)
{
    hipFree(gpuData);
}

// Cublas
void matmul_cublas(float* xout, float* x, float* w, float* bias, float *d_B, float *d_C, int n, int d)
{
    dim3 dimsA(n, d, 1);
    dim3 dimsB(1, n, 1);
    dim3 dimsC(dimsB.x, dimsA.y, 1);

    //int mem_size_A = n*d*sizeof(float);
    int mem_size_B = n*sizeof(float);
    int mem_size_C = d*sizeof(float);

    // copy host memory to device
    hipMemcpyAsync(d_B, x, mem_size_B, hipMemcpyHostToDevice);

    float beta = 0.0f;

    if(bias != NULL)
    {
        hipMemcpyAsync(d_C, bias, mem_size_C, hipMemcpyHostToDevice);
        beta = 1.0f;
    }

    // Calculate with Cublas
    const float alpha = 1.0f;

    hipblasStatus_t status = hipblasSgemm(
        handle, HIPBLAS_OP_T, HIPBLAS_OP_T, dimsA.y, dimsB.x,
        dimsA.x, &alpha, w, dimsA.x, d_B,
        dimsB.x, &beta, d_C, dimsC.y);

    // Copy result from device to host
    hipMemcpyAsync(xout, d_C, mem_size_C, hipMemcpyDeviceToHost);
}
